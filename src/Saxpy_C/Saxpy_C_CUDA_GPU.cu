#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


__global__ void saxpy(float *x, float *y, float a, long long size )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
        y[i] = a * x[i] + y[i];
}

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return (double)ts.tv_sec + (double)ts.tv_nsec / 1000000000.0;
}

int main() {

    int n = 29;
    int iterations = 1;
    float a = 3.1415f;

    double timings[n][iterations];

    for (int elements = 1; elements <= n; elements++) {

        long long size = 1 << elements;

        for (int it = 0; it < iterations; it++) {

            float* x = (float*)malloc(size * sizeof(float));
            float* y = (float*)malloc(size * sizeof(float));
            //float* z = (float*)malloc(size * sizeof(float));

            srand(time(NULL));

            for (long long i = 0; i < size; i++) {
                x[i] = ((float)rand()/(float)(RAND_MAX/10));
                y[i] = ((float)rand()/(float)(RAND_MAX/10));
            }


            //printf("%f\n", x[0]);
            //printf("\n");
            //printf("%f\n", y[0]);
            //printf("\n");

            float* d_x, *d_y;
            hipMalloc(&d_x, size * sizeof(float));
            hipMalloc(&d_y, size * sizeof(float));

            hipMemcpy(d_x, x, size * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(d_y, y, size * sizeof(float), hipMemcpyHostToDevice);

            int threadsPerBlock = 1024;
            int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;


            double start_time = get_time();
            saxpy<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, a, size);

            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                printf("CUDA error: %s\n", hipGetErrorString(err));
                exit(1);
            }
            
            hipDeviceSynchronize();
            double end_time = get_time();

            hipMemcpy(y, d_y, size * sizeof(float), hipMemcpyDeviceToHost);

           
            //printf("%f\n", y[0]);
            //printf("\n");


            double elapsed= end_time - start_time;
            //printf("%.15lf ", elapsed);

            //double elapsed = ((double)(end - start) / CLOCKS_PER_SEC) * 1000000;
            timings[elements-1][it] = elapsed;

            free(x);
            free(y);
            hipFree(d_x);
            hipFree(d_y);
            //free(z);
        }
    }

    double Average_Time[n];

    for (int i = 0; i < n; i++) {
        Average_Time[i] = 0;
    }

    for (int z = 0; z < n; z++) {
        printf("z = %d: ", z+1);
        for (int it = 0; it < iterations; it++) {
            Average_Time[z] += timings[z][it];
            printf("%.15lf ", timings[z][it]);
        }
        printf("\n");
    }

    for (int z = 0; z < n; z++) {
        printf("Average time for size %d: %.15lf seconds\n", z, Average_Time[z]/iterations);
    }

    FILE *outfile = fopen("Benchmark_Results/Saxpy_C_CUDA_GPU.csv", "w");

    for (int z = 0; z < n; z++) {
        fprintf(outfile, "%.15lf ", Average_Time[z]/iterations);
    }

    fclose(outfile);

    return 0;
}
