#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define BLOCK_SIZE 1024
#define N 536870912

__global__ void saxpy(float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        y[i] = a * x[i] + y[i];
}

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return (double)ts.tv_sec + (double)ts.tv_nsec / 1000000000.0;
}

int main(void)
{
    float *x, *y;
    float a = 2.0f;
    size_t size = N * sizeof(float);

    // Allocate memory for input and output vectors
    hipMallocManaged(&x, size);
    hipMallocManaged(&y, size);

    // Initialize input vectors
    for (int i = 0; i < N; ++i)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Launch kernel on the GPU and measure time

    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    double start_time = get_time();
    saxpy<<<blocksPerGrid, threadsPerBlock>>>(a, x, y);
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    double end_time = get_time();

    double elapsed= end_time - start_time;

    // Print results and timing information
    printf("Elapsed time: %.6f seconds\n", elapsed);
    for (int i = 0; i < 10; ++i)
        printf("y[%d] = %f\n", i, y[i]);

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}
